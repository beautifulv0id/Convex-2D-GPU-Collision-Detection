#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream> 
#include <fstream>
#include <cmath>
#include <math.h>
#include <random>
#include <chrono>

#include <npy.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

 /*
 This file creates a dataset of collision probabilities between a robot and an obstacle modeled as rectangles for different configurations using Monte Carlo sampling.
 One data-point defines the width, height and variance of the obstacle, as well as the position and orientation of the robot w.r.t the obstacle coordinate frame.
 First, NUM_POSES poses and NUM_VARIANCES variances are unifomely sampled from user-defined bounds. A pose contains the width and height of the obstacle and angle 
 theta of the robot. Variances are defined for the position and orientation of the obstacle. Finally, for each data-point a robot position is uniformly sampled, 
 as well as a random pose and variance from the pregenerated poses and variances. 
 */

#define N 4000 // numer of monte carlo smaples per iteration
#define MAX_RESAMPLE 1000
#define R_WIDTH 4.07 // width of the robot
#define R_HEIGHT 1.74 // height of the robot

#define NUM_BATCH 16777216 // number of configurations that are generated per batch
#define NUM_BATCHES 100 // number of batches 
#define NUM_DATA_POINTS double(NUM_BATCH)*NUM_BATCHES

#define NUM_POSES 64*64*64 // number of poses that are sampled, a pose contains the width, height of the obstacle and angle theta of the robot
#define NUM_VARIANCES 64*64*64*4 // number of variances that are sampled, 

#define POS_MIN -6.0 // minimium x-, y-position of the robot
#define POS_MAX 6.0 // maximum x-, y-position of the robot

#define OBSTACLE_WIDTH_MIN 0.1 // minimum width, height of obstacles
#define OBSTACLE_WIDTH_MAX 5.0 // maximum width, height of obstacles

#define VAR_MIN 0.001 // minimum positional and rotational variance 
#define VAR_MAX 0.3 // maximum positional and rotational variance 

#define N_ACCURACY_BINS 3

#define THREADS 1024
#define BLOCKS (int) ceil(NUM_BATCH/(float) THREADS)

#define DATA_DIR "data"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

struct Position
{
    float x, y;
};

struct Variance
{
    float x, y, theta, width, height;
};

struct Pose
{
    float width, height, theta;
};


typedef Variance StdDev;
typedef thrust::device_vector<Position>::iterator   DeviceFloatPairIterator;
typedef thrust::device_vector<float>::iterator   DeviceFloatIterator;
typedef thrust::device_vector<int>::iterator   DeviceIntIterator;
typedef thrust::tuple<DeviceFloatPairIterator, DeviceFloatIterator, DeviceFloatIterator, DeviceFloatIterator> DeviceIteratorTuple;
typedef thrust::zip_iterator<DeviceIteratorTuple> DeviceZipIterator;

void write_config(){
    std::ofstream confFile;
    confFile.open (DATA_DIR + std::string("/config.txt"), std::ios::out);
    confFile << "N" << "\t" << N << "\n";
    confFile << "R_WIDTH" << "\t" << R_WIDTH << "\n";
    confFile << "R_HEIGHT" << "\t" << R_HEIGHT << "\n";
    confFile << "NUM_BATCH" << "\t" << NUM_BATCH << "\n";
    confFile << "NUM_BATCHES" << "\t" << NUM_BATCHES << "\n";
    confFile << "NUM_DATA_POINTS" << "\t" << NUM_DATA_POINTS << "\n";
    confFile << "NUM_POSES" << "\t" << NUM_POSES << "\n";
    confFile << "NUM_VARIANCES" << "\t" << NUM_VARIANCES << "\n";
    confFile << "POS_MIN" << "\t" << POS_MIN << "\n";
    confFile << "POS_MAX" << "\t" << POS_MAX << "\n";
    confFile << "OBSTACLE_WIDTH_MIN" << "\t" << OBSTACLE_WIDTH_MIN << "\n";
    confFile << "OBSTACLE_WIDTH_MAX" << "\t" << OBSTACLE_WIDTH_MAX << "\n";
    confFile << "VAR_MIN" << "\t" << VAR_MIN << "\n";
    confFile << "VAR_MAX" << "\t" << VAR_MAX << "\n";
    confFile.close();

}
__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(7+id, id, 0, &state[id]);
}

__device__ __host__
void create_rect(float* r, float w, float h)
{
    r[0] = -w/2;
    r[1] = -h/2;
    r[2] = w/2;
    r[3] = -h/2;
    r[4] = w/2;
    r[5] = h/2;
    r[6] = -w/2;
    r[7] = h/2; 
}

__device__ void rot_trans_rectangle(float* r, float dx, float dy, float dt){
    float c = cosf(dt);
    float s = sinf(dt);
    float x,y;
    for(int i = 0; i < 4; i++){
        x = r[2*i];
        y = r[2*i+1];
        r[2*i] = c*x-s*y + dx;
        r[2*i+1] = s*x+c*y + dy;
    }
}

__device__ void sample_rectangle(float* r_in, float* r_out, StdDev& std_dev, hiprandState* state)
{
    float dx = hiprand_normal(state) * std_dev.x;
    float dy = hiprand_normal(state) * std_dev.y; 
    float dt = hiprand_normal(state) * std_dev.theta;
    float dw = hiprand_normal(state) * std_dev.width; 
    float dh = hiprand_normal(state) * std_dev.height;
    
    memcpy(r_out, r_in, sizeof(float) * 8);
    float dwh[8];
    create_rect(dwh, dw, dh);
    for (int i = 0; i < 8; i++){ r_out[i] += dwh[i]; }
    rot_trans_rectangle(r_out, dx, dy, dt);
}

__device__ int convex_collide(float* r1, float* r2) {
    float* rs[2] = {r1, r2};
    float norm[2];
    float p1[4];
    float p2[4];
    float* r;

    int collide = 1;
    for(int j = 0; j < 2; j++){
        r = rs[j];
        for(int i = 0; i < 4; i++){
            norm[0] = r[(i+1)*2%8] - r[i*2];
            norm[1] = r[((i+1)*2+1)%8] - r[i*2+1];
            for(int k = 0; k < 4; k++){
                p1[k] = norm[0]*r1[k*2]+norm[1]*r1[k*2+1];
                p2[k] = norm[0]*r2[k*2]+norm[1]*r2[k*2+1];
            }
            thrust::pair<float *, float *> result1 = thrust::minmax_element(thrust::device, p1, p1 + 4);
            thrust::pair<float *, float *> result2 = thrust::minmax_element(thrust::device, p2, p2 + 4);
            if(*result1.second < *result2.first || *result2.second < *result1.first){
                collide = 0;
            }
        }
    }
    return collide;
}

__device__
float calcSlack(int nsamples, int nsamples_true){
    float z = 1.96;
    float alpha = 0.025;
    if((nsamples_true == nsamples) || (nsamples_true == 0)){
        return log(1.0 / alpha) / nsamples;
    }
    else{
        return z / nsamples * sqrt((float) nsamples_true - nsamples_true * nsamples_true / (float) nsamples);
    }
}


__global__ void monte_carlo_sample_collision_dataset_uniform(float* robot_base,
                                                                Pose* poses,
                                                                StdDev* std_devs,
                                                                float* pose_idxs,
                                                                float* std_dev_idxs,
                                                                Position* positions,
                                                                float* cps,
                                                                float* accuracy_bins,
                                                                float* bin_slack,
                                                                int* done,
                                                                int iteration,
                                                                int num_left,
                                                                hiprandState* state) {
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(gidx >= num_left)
        return;

    hiprandState* localState = &state[gidx];

    int pose_idx;
    int std_dev_idx;
    Position pos;
    int n_samplestrue = 0;
    float cp_out;
    if(iteration == 0){
        // pos.x = hiprand_normal(localState) * 3.9/1.96;
        // pos.y = hiprand_normal(localState) * 3.9/1.96;
        pos.x = POS_MIN + hiprand_uniform(localState) * (POS_MAX-POS_MIN);
        pos.y = POS_MIN + hiprand_uniform(localState) * (POS_MAX-POS_MIN);
        pose_idx = hiprand(localState) % NUM_POSES;
        std_dev_idx = hiprand(localState) % NUM_VARIANCES;
    } else {
        n_samplestrue = (int) cps[gidx];
        pos = positions[gidx];
        pose_idx = pose_idxs[gidx];
        std_dev_idx = std_dev_idxs[gidx];
    }

    Pose pose = poses[pose_idx];
    StdDev std_dev = std_devs[std_dev_idx];

    float obstacle[8];
    create_rect(obstacle, pose.width, pose.height);    
    float sampled_obstacle[8];
    float robot[8];

    memcpy(robot, robot_base, sizeof(float) * 8);
    rot_trans_rectangle(robot, pos.x, pos.y, pose.theta);

    for (int i = 0; i < N; i++)
    {
        sample_rectangle(obstacle, sampled_obstacle, std_dev, localState);
        n_samplestrue += convex_collide(robot, sampled_obstacle);
    }
    int n_samples = N * (iteration+1);
    float slack = calcSlack(n_samples, n_samplestrue);

    float p = (float) n_samplestrue / (float) n_samples;
    int d = 0;
    if(iteration+1 == MAX_RESAMPLE){
        printf("n_samplestrue %i, n_samples %i, %f\n", n_samplestrue, n_samples, slack);
        d = 1;
    }
    else {
        for (int i = 0; i < N_ACCURACY_BINS; i++){
            // a bit hacky, but actually results in accuracy_bins[i] <=  p < accuracy_bins[i+1] (which is what we want)
            if(p >= accuracy_bins[i] && p <= accuracy_bins[i+1] && slack < bin_slack[i]){
                d = 1;
            }
        }
    }

    cp_out = n_samplestrue;
    if(d){
        cp_out = p;
    }

    done[gidx] = d;
    cps[gidx] = cp_out;
    positions[gidx] = pos;
    pose_idxs[gidx] = pose_idx;
    std_dev_idxs[gidx] = std_dev_idx;
}

void print_help(int* done, int* d_done, int* index, int* d_index){
    hipMemcpy(done, d_done, sizeof(int) * NUM_BATCH, hipMemcpyDeviceToHost);
    hipMemcpy(index, d_index, sizeof(int) * NUM_BATCH, hipMemcpyDeviceToHost);
    printf("done:  ");
    for (size_t i = 0; i < NUM_BATCH; i++)
    {
        printf("%2d ", done[i]);
    }
    printf("\n");
    printf("index: ");
    for (size_t i = 0; i < NUM_BATCH; i++)
    {
        printf("%2d ", index[i]);
    }
    printf("\n");
}

int main(){
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);

    for (int i = 0; i < 1; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
                prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
                prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
        printf("  maxThreadsDim.x: %i\n", prop.maxThreadsDim[0]);
        printf("  maxThreadsDim.y: %i\n", prop.maxThreadsDim[1]);        
        printf("  maxThreadsDim.z: %i\n", prop.maxThreadsDim[2]);
        printf("  maxThreadsPerBlock: %i\n", prop.maxThreadsPerBlock);
        printf("  maxBlocksPerMultiProcessor: %i\n", prop.maxBlocksPerMultiProcessor);
        printf("  maxGridSize.x: %i\n", prop.maxGridSize[0]);
        printf("  maxGridSize.y: %i\n", prop.maxGridSize[1]);        
        printf("  maxGridSize.z: %i\n", prop.maxGridSize[2]);

    }

    write_config();

    Pose* poses = (Pose*) malloc(sizeof(Pose)*NUM_POSES);
    StdDev* std_devs = (Variance*) malloc(sizeof(StdDev)*NUM_VARIANCES);  
    std::vector<Variance> variances(NUM_VARIANCES);  

    std::default_random_engine generator;
    auto obstacle_uniform = std::uniform_real_distribution<float>(OBSTACLE_WIDTH_MIN, OBSTACLE_WIDTH_MAX);
    auto obstacle_scale_uniform = std::uniform_real_distribution<float>(0.25, 1.);
    auto theta_uniform = std::uniform_real_distribution<float>(0.0, 2.0*M_PI);
    auto variance_uniform = std::uniform_real_distribution<float>(VAR_MIN, VAR_MAX);
    
    for (int i = 0; i < NUM_POSES; i++)
    {
        poses[i].width = obstacle_uniform(generator);
        poses[i].height = obstacle_uniform(generator);
        poses[i].theta = theta_uniform(generator);
    }    
    for (int i = 0; i < NUM_VARIANCES; i++)
    {
        variances[i].x = variance_uniform(generator);
        variances[i].y = variance_uniform(generator);
        variances[i].theta = variance_uniform(generator);       
        variances[i].width = variance_uniform(generator);       
        variances[i].height = variance_uniform(generator);       
        std_devs[i].x = sqrt(variances[i].x);
        std_devs[i].y = sqrt(variances[i].y);
        std_devs[i].theta = sqrt(variances[i].theta);   
        std_devs[i].width = sqrt(variances[i].width);   
        std_devs[i].height = sqrt(variances[i].height);   
    }

    // write poses and variances
    size_t poses_shape[2] = {(size_t) NUM_POSES, (size_t) 3};
    size_t variances_shape[2] = {(size_t) NUM_VARIANCES, (size_t) 5};
    npy::SaveArrayAsNumpy(DATA_DIR + std::string("/poses.npy"), false, 2, poses_shape, (float*) poses);
    npy::SaveArrayAsNumpy(DATA_DIR + std::string("/variances.npy"), false, 2, variances_shape, (float*) variances.data());

    float accuracy_bins[4] = {0, 0.01, 0.1, 1.};
    float bin_slack[4] = {0.0001, 0.005, 0.01, 0};

    float* robot = (float*) malloc(sizeof(float)*4*2);
    Position* positions = (Position*) malloc(sizeof(Position) * NUM_BATCH);
    float* pose_idxs = (float*) malloc(sizeof(float) * NUM_BATCH);
    float* var_idxs = (float*) malloc(sizeof(float) * NUM_BATCH);
    float* cp = (float*) malloc(sizeof(float) * NUM_BATCH);
    
    float* d_accuracy_bins;
    float* d_bin_slack;
    float* d_robot; 
    Pose* d_poses; 
    StdDev* d_std_devs; 
    Position* d_positions; 
    float* d_pose_idxs; 
    float* d_var_idxs; 
    float* d_cp; 
    int* d_done; 

    hipMalloc(&d_accuracy_bins, sizeof(float)*(4));
    hipMalloc(&d_bin_slack, sizeof(float)*(4));
    hipMalloc(&d_robot, sizeof(float)*(4*2));
    hipMalloc(&d_poses, sizeof(Pose)*NUM_POSES);
    hipMalloc(&d_std_devs, sizeof(StdDev)*NUM_VARIANCES);
    hipMalloc(&d_positions, sizeof(Position)*NUM_BATCH);
    hipMalloc(&d_pose_idxs, sizeof(float)*(NUM_BATCH));
    hipMalloc(&d_var_idxs, sizeof(float)*(NUM_BATCH));
    hipMalloc(&d_cp, sizeof(float)*(NUM_BATCH));
    hipMalloc(&d_done, sizeof(int)*(NUM_BATCH));

    DeviceZipIterator d_iter(thrust::make_tuple(thrust::device_pointer_cast(d_positions), 
                                        thrust::device_pointer_cast(d_cp),
                                        thrust::device_pointer_cast(d_var_idxs),
                                        thrust::device_pointer_cast(d_pose_idxs)));


    std::vector<float> dataset(NUM_BATCH*5);

    hiprandState *devStates;
    hipMalloc((void **)&devStates, NUM_BATCH *  sizeof(hiprandState));

    dim3 threadsPerBlock(1024);
    dim3 numBlocks((int) ceil(NUM_BATCH/threadsPerBlock.x));  

    // Initialize array
    create_rect(robot, R_WIDTH, R_HEIGHT);

    // Transfer data from host to device memory
    hipMemcpy(d_robot, robot, sizeof(float)*(4*2), hipMemcpyHostToDevice);
    hipMemcpy(d_poses, poses, sizeof(Pose)*NUM_POSES, hipMemcpyHostToDevice);
    hipMemcpy(d_std_devs, std_devs, sizeof(Variance)*NUM_VARIANCES, hipMemcpyHostToDevice);
    hipMemcpy(d_accuracy_bins, accuracy_bins, sizeof(float)*(4), hipMemcpyHostToDevice);
    hipMemcpy(d_bin_slack, bin_slack, sizeof(float)*(4), hipMemcpyHostToDevice);
    setup_kernel<<<BLOCKS, THREADS>>>(devStates);
    CUDA_CALL(hipPeekAtLastError());

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    std::cout << "Total number of configurations: " << NUM_BATCH << std::endl;
    std::cout << "Begin computation..." << std::endl;
    int counter = 0;
    printf("batches generated: %i/%i\n", counter, NUM_BATCHES);

    for (int batch_index = 0; batch_index < NUM_BATCHES; batch_index++)
    {
        int num_left = NUM_BATCH;
        int batch_done = 0;
        int iteration = 0;
        while(num_left != 0 && iteration < MAX_RESAMPLE){
            numBlocks = ((int) ceil((float) num_left/threadsPerBlock.x));  
            monte_carlo_sample_collision_dataset_uniform<<<numBlocks, threadsPerBlock>>>(
                d_robot,
                d_poses,
                d_std_devs,
                d_pose_idxs,
                d_var_idxs,
                d_positions,
                d_cp,
                d_accuracy_bins,
                d_bin_slack,
                d_done,
                iteration,
                num_left,
                devStates
            );
            batch_done = thrust::count(thrust::device, thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + num_left), 1);
            if(batch_done > 0){
                thrust::sort_by_key(thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + num_left), d_iter);
                num_left -= batch_done;
                hipMemcpy(positions + num_left, d_positions + num_left, sizeof(Position) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(cp + num_left, d_cp + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(var_idxs + num_left, d_var_idxs + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(pose_idxs + num_left, d_pose_idxs + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
            }
            iteration++;
            printf("num left %i\n", num_left);
        }

        if(num_left > 0){
            printf("copying remaining %i over (0, %i)\n", num_left, num_left);
            hipMemcpy(positions, d_positions, sizeof(Position) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(cp, d_cp, sizeof(float) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(var_idxs, d_var_idxs, sizeof(float) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(pose_idxs, d_pose_idxs, sizeof(float) * num_left, hipMemcpyDeviceToHost); 
        }


        CUDA_CALL(hipDeviceSynchronize());
        printf("\33[2K\r");
        printf("batches generated: %i/%i", ++counter, NUM_BATCHES);
        printf("num left %i", num_left);
        fflush(stdout); 

        // write data
        float* d = dataset.data();
        float* c = cp;

        for (int j = 0; j < NUM_BATCH; j++)
        {
            d[0] = positions[j].x;  // x
            d[1] = positions[j].y;  // y
            d[2] = *c;  // cp
            d[3] = var_idxs[j]; // var_idx
            d[4] = pose_idxs[j]; // pose_idx
            c+=1;
            d+=5;
        }

        // write dataset
        size_t ds_shape[2] = {(size_t) NUM_BATCH, (size_t) 5};

        npy::SaveArrayAsNumpy(DATA_DIR + std::string("/") + std::to_string(batch_index) + ".npy", false, 2, ds_shape, dataset);
    }
    std::cout << std::endl;
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Finished computation" << std::endl;
    std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " [ms]" << std::endl;

    // free memory
    hipFree(devStates);
    hipFree(d_accuracy_bins);
    hipFree(d_bin_slack);
    hipFree(d_robot);
    hipFree(d_poses);
    hipFree(d_std_devs);
    hipFree(d_positions);
    hipFree(d_pose_idxs);
    hipFree(d_var_idxs);
    hipFree(d_cp);
    hipFree(d_done);
    std::cout << "Done." << std::endl;
}
