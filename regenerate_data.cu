#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream> 
#include <fstream>
#include <cmath>
#include <math.h>
#include <random>
#include <chrono>
#include<ctime>

#include <npy.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <cstddef>

#include <algorithm>    

 /*
 This file creates a dataset of collision probabilities between a robot and an obstacle modeled as rectangles for different configurations using Monte Carlo sampling.
 One data-point defines the width, height and variance of the obstacle, as well as the position and orientation of the robot w.r.t the obstacle coordinate frame.
 First, NUM_POSES poses and NUM_VARIANCES variances are unifomely sampled from user-defined bounds. A pose contains the width and height of the obstacle and angle 
 theta of the robot. Variances are defined for the position and orientation of the obstacle. Finally, for each data-point a robot position is uniformly sampled, 
 as well as a random pose and variance from the pregenerated poses and variances. 
 */

#define UNIFORM_SAMPLING 0

#define N 4000000

// SIMULATION
#define R_WIDTH 4.07 // width of the robot
#define R_HEIGHT 1.74 // height of the robot

// TIAGO
// #define R_WIDTH 0.75 // width of the robot
// #define R_HEIGHT 0.8 // height of the robot


#define R_OFFSET ((R_WIDTH + R_HEIGHT) / 4)

#define POS_MIN -12.0 // minimium x-, y-position of the robot
#define POS_MAX 12.0 // maximum x-, y-position of the robot

#define OBSTACLE_WIDTH_MIN 0.1 // minimum width, height of obstacles
#define OBSTACLE_WIDTH_MAX 5.0 // maximum width, height of obstacles

#define VAR_MIN 0.0 // minimum positional and rotational variance 
#define VAR_MAX 0.3 // maximum positional and rotational variance 


#define THREADS 512
 
// float accuracy_bins[] = {0, 0.001, 0.01, 0.1, 1};
// float bin_slack[] = {0.00005,0.0005, 0.001, 0.01, 0};
#define N_ACCURACY_BINS 3
float accuracy_bins[N_ACCURACY_BINS+1] = {0, 0.01, 0.1, 1};
float bin_slack[N_ACCURACY_BINS+1] = {0.0001, 0.001, 0.01, 0};
// float bin_slack[] = {0.00002,0.0001, 0.0005, 0.001, 0};



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

struct Position
{
    float x, y;
};

struct Variance
{
    float x, y, theta, width, height;
};

struct Pose
{
    float width, height, theta;
};


typedef Variance StdDev;
typedef thrust::device_vector<Position>::iterator   DeviceFloatPairIterator;
typedef thrust::device_vector<float>::iterator   DeviceFloatIterator;
typedef thrust::device_vector<int>::iterator   DeviceIntIterator;
typedef thrust::tuple<DeviceFloatPairIterator, DeviceFloatIterator, DeviceFloatIterator, DeviceFloatIterator> DeviceIteratorTuple;
typedef thrust::zip_iterator<DeviceIteratorTuple> DeviceZipIterator;

void write_config(std::string data_in, int num_batch, int num_batches){
    std::ofstream confFile;
    confFile.open (data_in + std::string("/config.txt"), std::ios::out);
    confFile << "UNIFORM_SAMPLING" << "\t" << UNIFORM_SAMPLING << "\n";
    confFile << "N" << "\t" << N << "\n";
    confFile << "R_WIDTH" << "\t" << R_WIDTH << "\n";
    confFile << "R_HEIGHT" << "\t" << R_HEIGHT << "\n";
    confFile << "NUM_BATCH" << "\t" << num_batch << "\n";
    confFile << "NUM_BATCHES" << "\t" << num_batches << "\n";
    confFile << "NUM_DATA_POINTS" << "\t" << num_batch * num_batches << "\n";
    confFile << "POS_MIN" << "\t" << POS_MIN << "\n";
    confFile << "POS_MAX" << "\t" << POS_MAX << "\n";
    confFile << "OBSTACLE_WIDTH_MIN" << "\t" << OBSTACLE_WIDTH_MIN << "\n";
    confFile << "OBSTACLE_WIDTH_MAX" << "\t" << OBSTACLE_WIDTH_MAX << "\n";
    confFile << "VAR_MIN" << "\t" << VAR_MIN << "\n";
    confFile << "VAR_MAX" << "\t" << VAR_MAX << "\n";
    confFile << "ACCURACY_BINS" << "\t";
    for (size_t i = 0; i < N_ACCURACY_BINS+1; i++)
        confFile << accuracy_bins[i] << " ";
    confFile << "\n";
    confFile << "BIN_SLACK" << "\t";
    for (size_t i = 0; i < N_ACCURACY_BINS; i++)
        confFile << bin_slack[i] << " ";
    confFile << "\n";
    
    confFile.close();

}
__global__ void setup_kernel(hiprandState *state, int seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(seed+id, id, 0, &state[id]);
}

__device__ __host__
void create_rect(float* r, float w, float h)
{
    r[0] = -w/2;
    r[1] = -h/2;
    r[2] = w/2;
    r[3] = -h/2;
    r[4] = w/2;
    r[5] = h/2;
    r[6] = -w/2;
    r[7] = h/2; 
}

__device__ void rot_trans_rectangle(float* r, float dx, float dy, float dt){
    float c = cosf(dt);
    float s = sinf(dt);
    float x,y;
    for(int i = 0; i < 4; i++){
        x = r[2*i];
        y = r[2*i+1];
        r[2*i] = c*x-s*y + dx;
        r[2*i+1] = s*x+c*y + dy;
    }
}

__device__ void sample_rectangle(float* r_in, float* r_out, StdDev& std_dev, hiprandState* state)
{
    float dx = hiprand_normal(state) * std_dev.x;
    float dy = hiprand_normal(state) * std_dev.y; 
    float dt = hiprand_normal(state) * std_dev.theta;
    float dw = hiprand_normal(state) * std_dev.width; 
    float dh = hiprand_normal(state) * std_dev.height;
    
    memcpy(r_out, r_in, sizeof(float) * 8);
    float dwh[8];
    create_rect(dwh, dw, dh);
    for (int i = 0; i < 8; i++){ r_out[i] += dwh[i]; }
    rot_trans_rectangle(r_out, dx, dy, dt);
}

__device__ int convex_collide(float* r1, float* r2) {
    float* rs[2] = {r1, r2};
    float norm[2];
    float p1[4];
    float p2[4];
    float* r;

    int collide = 1;
    for(int j = 0; j < 2; j++){
        r = rs[j];
        for(int i = 0; i < 4; i++){
            norm[0] = r[(i+1)*2%8] - r[i*2];
            norm[1] = r[((i+1)*2+1)%8] - r[i*2+1];
            for(int k = 0; k < 4; k++){
                p1[k] = norm[0]*r1[k*2]+norm[1]*r1[k*2+1];
                p2[k] = norm[0]*r2[k*2]+norm[1]*r2[k*2+1];
            }
            thrust::pair<float *, float *> result1 = thrust::minmax_element(thrust::device, p1, p1 + 4);
            thrust::pair<float *, float *> result2 = thrust::minmax_element(thrust::device, p2, p2 + 4);
            if(*result1.second < *result2.first || *result2.second < *result1.first){
                collide = 0;
            }
        }
    }
    return collide;
}

__device__
float calcSlack(int nsamples, int nsamples_true){
    float z = 1.96;
    float alpha = 0.025;
    if((nsamples_true == nsamples) || (nsamples_true == 0)){
        return log(1.0 / alpha) / nsamples;
    }
    else{
        return z / nsamples * sqrt((float) nsamples_true - nsamples_true * nsamples_true / (float) nsamples);
    }
}

__device__
int getBin(float p, float* accuracy_bins){
    int bin = 0;
    for (int i = 0; i < N_ACCURACY_BINS; i++){
        if(p >= accuracy_bins[i] && p <= accuracy_bins[i+1]){
            bin = i;
        }
    }
    return bin;
}


__global__ void write_collision_probability(float* n_true, int n_done, int n_samples){
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(gidx >= n_done)
        return;
    n_true[gidx] =  n_true[gidx] / (float) n_samples;
}


__global__ void monte_carlo_sample_collision_dataset_uniform(float* robot_base,
                                                                Pose* poses,
                                                                StdDev* std_devs,
                                                                float* pose_idxs,
                                                                float* std_dev_idxs,
                                                                Position* positions,
                                                                float* cps,
                                                                float* accuracy_bins,
                                                                float* bin_slack,
                                                                int* done,
                                                                int iteration,
                                                                int n_samples,
                                                                int n_batch,
                                                                int num_left,
                                                                hiprandState* state) {
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(gidx >= num_left)
        return;

    hiprandState* localState = &state[gidx];

    int pose_idx;
    int std_dev_idx;
    Position pos;
    int n_samplestrue = 0;
    Pose pose;
    StdDev std_dev;

    n_samplestrue = (int) cps[gidx];
    pos = positions[gidx];
    pose_idx = pose_idxs[gidx];
    std_dev_idx = std_dev_idxs[gidx];
    pose = poses[pose_idx];
    std_dev = std_devs[std_dev_idx];


    float obstacle[8];
    create_rect(obstacle, pose.width, pose.height);    
    float sampled_obstacle[8];
    float robot[8];

    memcpy(robot, robot_base, sizeof(float) * 8);
    rot_trans_rectangle(robot, pos.x, pos.y, pose.theta);

    for (int i = 0; i < n_batch; i++)
    {
        sample_rectangle(obstacle, sampled_obstacle, std_dev, localState);
        n_samplestrue += convex_collide(robot, sampled_obstacle);
    }
    float slack = calcSlack(n_samples, n_samplestrue);

    float p = (float) n_samplestrue / (float) n_samples;
    int d = 0;
    if(slack <= bin_slack[getBin(p, accuracy_bins)]){
        d = 1;
    }

    done[gidx] = d;
    cps[gidx] = n_samplestrue;
}

int main(int argc, char* argv[])
{   
    std::string data_in = "data_in";
    std::string data_out = "data_out";
    int start_batch_count = 0;
    int num_batches = 100;
    if(argc > 1){
        data_in = std::string(argv[1]);
        if(argc > 2)
            data_out = std::string(argv[2]);
        if (argc > 3) 
            num_batches = std::stoi(argv[3]);
        if(argc > 4)
            start_batch_count = std::stoi(argv[4]);
    }
    
    std::cout << "data dir: " << data_in << std::endl;
    std::cout << "num batches: " << num_batches << std::endl;
    std::cout << "start batch count: " << start_batch_count << std::endl;

    std::vector<double> pos_with_idx_Loader;
    std::vector<float> posesLoader;
    std::vector<float> variancesLoader;

    std::vector<npy::ndarray_len_t> poses_shape;
    std::vector<npy::ndarray_len_t> variances_shape;
    std::vector<npy::ndarray_len_t> pos_with_idx_shape;

    npy::LoadArrayFromNumpy(data_in + std::string("/poses.npy"), poses_shape, posesLoader);
    npy::LoadArrayFromNumpy(data_in + std::string("/variances.npy"), variances_shape, variancesLoader);
    npy::LoadArrayFromNumpy(data_in + std::string("/0.npy"), pos_with_idx_shape, pos_with_idx_Loader);


    int num_poses = poses_shape[0];
    int num_variances = variances_shape[0];
    int num_data_points = pos_with_idx_shape[0] / 4;

    std::cout << "num poses: " << num_poses << std::endl;
    std::cout << "num variances: " << num_variances << std::endl;
    std::cout << "num data points: " << num_data_points << std::endl;
    
    Pose* poses = (Pose*) malloc(sizeof(Pose)*num_poses);
    StdDev* std_devs = (Variance*) malloc(sizeof(StdDev)*num_variances);  
    std::vector<Variance> variances(num_variances);  
    Position* positions = (Position*) malloc(sizeof(Position) * num_data_points);

    float* var_idxs = (float*) malloc(sizeof(float) * num_data_points);
    float* pose_idxs = (float*) malloc(sizeof(float) * num_data_points);

    for (int i = 0; i < num_poses; i++)
    {
        poses[i].width = posesLoader[i*3];
        poses[i].height = posesLoader[i*3+1];
        poses[i].theta = posesLoader[i*3+2];
    }

    for (int i = 0; i < num_variances; i++){
        variances[i].x = variancesLoader[i*5];
        variances[i].y = variancesLoader[i*5+1];
        variances[i].theta = variancesLoader[i*5+2];
        variances[i].width = variancesLoader[i*5+3];
        variances[i].height = variancesLoader[i*5+4];
        std_devs[i].x = sqrt(variances[i].x);
        std_devs[i].y = sqrt(variances[i].y);
        std_devs[i].theta = sqrt(variances[i].theta);
        std_devs[i].width = sqrt(variances[i].width);
        std_devs[i].height = sqrt(variances[i].height);
    }

    size_t accuracy_bins_shape[1] = {(size_t) (sizeof(accuracy_bins)/sizeof(accuracy_bins[0]))};
    size_t bin_slack_shape[1] = {(size_t) (sizeof(bin_slack)/sizeof(bin_slack[0]))};
    npy::SaveArrayAsNumpy(data_out + "/meta" + std::string("/accuracy_bins.npy"), false, 1, accuracy_bins_shape, accuracy_bins);
    npy::SaveArrayAsNumpy(data_out + "/meta" + std::string("/bin_slack.npy"), false, 1, bin_slack_shape, bin_slack);

    float* robot = (float*) malloc(sizeof(float)*4*2);
    float* cp = (float*) malloc(sizeof(float) * num_data_points);
    
    float* d_accuracy_bins;
    float* d_bin_slack;
    float* d_robot; 
    Pose* d_poses; 
    StdDev* d_std_devs; 
    Position* d_positions; 
    float* d_pose_idxs; 
    float* d_var_idxs; 
    float* d_cp; 
    int* d_done; 

    hipMalloc(&d_accuracy_bins, sizeof(float)*(N_ACCURACY_BINS+1));
    hipMalloc(&d_bin_slack, sizeof(float)*(N_ACCURACY_BINS+1));
    hipMalloc(&d_robot, sizeof(float)*(4*2));
    hipMalloc(&d_poses, sizeof(Pose)*num_poses);
    hipMalloc(&d_std_devs, sizeof(StdDev)*num_variances);
    hipMalloc(&d_positions, sizeof(Position)*num_data_points);
    hipMalloc(&d_pose_idxs, sizeof(float)*(num_data_points));
    hipMalloc(&d_var_idxs, sizeof(float)*(num_data_points));
    hipMalloc(&d_cp, sizeof(float)*(num_data_points));
    hipMalloc(&d_done, sizeof(int)*(num_data_points));

    DeviceZipIterator d_iter(thrust::make_tuple(thrust::device_pointer_cast(d_positions), 
                                        thrust::device_pointer_cast(d_cp),
                                        thrust::device_pointer_cast(d_var_idxs),
                                        thrust::device_pointer_cast(d_pose_idxs)));


    std::vector<float> dataset(num_data_points*5);

    hiprandState *devStates;
    hipMalloc((void **)&devStates, num_data_points *  sizeof(hiprandState));

    dim3 threadsPerBlock(THREADS);
    dim3 numBlocks((int) std::max(1.0, ceil(num_data_points/threadsPerBlock.x)));  

    // Initialize array
    create_rect(robot, R_WIDTH, R_HEIGHT);

    // Transfer data from host to device memory
    hipMemcpy(d_robot, robot, sizeof(float)*(4*2), hipMemcpyHostToDevice);
    hipMemcpy(d_poses, poses, sizeof(Pose)*num_poses, hipMemcpyHostToDevice);
    hipMemcpy(d_std_devs, std_devs, sizeof(Variance)*num_variances, hipMemcpyHostToDevice);
    hipMemcpy(d_accuracy_bins, accuracy_bins, sizeof(float)*(N_ACCURACY_BINS+1), hipMemcpyHostToDevice);
    hipMemcpy(d_bin_slack, bin_slack, sizeof(float)*(N_ACCURACY_BINS+1), hipMemcpyHostToDevice);
    // std::srand(std::time(0));

    setup_kernel<<<numBlocks, THREADS>>>(devStates, std::rand());

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    std::cout << "Total number of configurations: " << num_data_points * num_batches << std::endl;
    std::cout << "Begin computation..." << std::endl;
    int counter = 0;
    printf("batches generated: %i/%i\n", counter, num_batches);
    int t = thrust::count(thrust::device, thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + 20), 1);

    for (int batch_index = 0; batch_index < num_batches; batch_index++)
    {
        pos_with_idx_Loader.clear();
        npy::LoadArrayFromNumpy(data_in + std::string("/") + std::to_string(batch_index) + std::string(".npy"), pos_with_idx_shape, pos_with_idx_Loader);
        for (int i = 0; i < num_data_points; i++)
        {
            positions[i].x = (float) pos_with_idx_Loader[i*4];
            positions[i].y = (float) pos_with_idx_Loader[i*4+1];
            var_idxs[i] = (float) pos_with_idx_Loader[i*4+2];
            pose_idxs[i] = (float) pos_with_idx_Loader[i*4+3];
        }
        // upload positions, pose_idxs, var_idxs
        hipMemcpy(d_positions, positions, sizeof(Position) * num_data_points, hipMemcpyHostToDevice);
        hipMemcpy(d_pose_idxs, pose_idxs, sizeof(float) * num_data_points, hipMemcpyHostToDevice);
        hipMemcpy(d_var_idxs, var_idxs, sizeof(float) * num_data_points, hipMemcpyHostToDevice);
        gpuErrchk( hipPeekAtLastError() );
        int num_left = num_data_points;
        int batch_done = 0;
        int iteration = 0;
        int n_samples = 0;
        int n_batch = 0;
        while(num_left > 0 && n_samples < N){
            numBlocks = ((int) ceil((float) num_left/threadsPerBlock.x));  
            if(n_samples < 20000)
                n_batch = 1000;
            else
                n_batch = 100000;
            n_samples += n_batch;
            monte_carlo_sample_collision_dataset_uniform<<<numBlocks, threadsPerBlock>>>(
                d_robot,
                d_poses,
                d_std_devs,
                d_pose_idxs,
                d_var_idxs,
                d_positions,
                d_cp,
                d_accuracy_bins,
                d_bin_slack,
                d_done,
                iteration,
                n_samples,
                n_batch,
                num_left,
                devStates
            );
            gpuErrchk( hipPeekAtLastError() );
            batch_done = thrust::count(thrust::device, thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + num_left), 1);
            gpuErrchk( hipPeekAtLastError() );
            if(batch_done > 0){
                thrust::sort_by_key(thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + num_left), d_iter);
                num_left -= batch_done;
                numBlocks = ((int) ceil((float) batch_done/threadsPerBlock.x));  
                write_collision_probability<<<numBlocks, threadsPerBlock>>>(d_cp + num_left, batch_done, n_samples);
                hipMemcpy(positions + num_left, d_positions + num_left, sizeof(Position) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(cp + num_left, d_cp + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(var_idxs + num_left, d_var_idxs + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(pose_idxs + num_left, d_pose_idxs + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
            }
            iteration++;
        }

        if(num_left > 0){
            printf("copying remaining %i over (0, %i)\n", num_left, num_left);
            numBlocks = ((int) ceil((float) num_left/threadsPerBlock.x));  
            write_collision_probability<<<numBlocks, threadsPerBlock>>>(d_cp, num_left, n_samples);
            gpuErrchk( hipPeekAtLastError() );
            hipMemcpy(positions, d_positions, sizeof(Position) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(cp, d_cp, sizeof(float) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(var_idxs, d_var_idxs, sizeof(float) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(pose_idxs, d_pose_idxs, sizeof(float) * num_left, hipMemcpyDeviceToHost); 
        }


        CUDA_CALL(hipDeviceSynchronize());

        // write data
        float* d = dataset.data();
        float* c = cp;

        for (int j = 0; j < num_data_points; j++)
        {
            d[0] = positions[j].x;  // x
            d[1] = positions[j].y;  // y
            d[2] = *c;  // cp
            d[3] = var_idxs[j]; // var_idx
            d[4] = pose_idxs[j]; // pose_idx
            c+=1;
            d+=5;
        }

        // write dataset
        size_t ds_shape[2] = {(size_t) num_data_points, (size_t) 5};

        npy::SaveArrayAsNumpy(data_out + std::string("/") + std::to_string(start_batch_count + batch_index) + ".npy", false, 2, ds_shape, dataset);
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        printf("\33[2K\r");
        printf("batches generated: %i/%i, Time: %i [min]", ++counter, num_batches, (int) std::chrono::duration_cast<std::chrono::minutes>(end - begin).count());
        fflush(stdout); 
    }
    std::cout << std::endl;
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Finished computation" << std::endl;
    std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::minutes>(end - begin).count() << " [min]" << std::endl;

    // free memory
    hipFree(devStates);
    hipFree(d_accuracy_bins);
    hipFree(d_bin_slack);
    hipFree(d_robot);
    hipFree(d_poses);
    hipFree(d_std_devs);
    hipFree(d_positions);
    hipFree(d_pose_idxs);
    hipFree(d_var_idxs);
    hipFree(d_cp);
    hipFree(d_done);
    std::cout << "Done." << std::endl;
}
