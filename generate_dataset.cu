#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream> 
#include <fstream>
#include <cmath>
#include <math.h>
#include <random>
#include <chrono>
#include<ctime>

#include <npy.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <cstddef>

#include <algorithm>    
#include <boost/program_options.hpp>

 /*
 This file creates a dataset of collision probabilities between a robot and an obstacle modeled as rectangles for different configurations using Monte Carlo sampling.
 One data-point defines the width, height and variance of the obstacle, as well as the position and orientation of the robot w.r.t the obstacle coordinate frame.
 First, num_poses poses and num_variances variances are unifomely sampled from user-defined bounds. A pose contains the width and height of the obstacle and angle 
 theta of the robot. Variances are defined for the position and orientation of the obstacle. Finally, for each data-point a robot position is uniformly sampled, 
 as well as a random pose and variance from the pregenerated poses and variances. 
 */

namespace po = boost::program_options;

struct Arguments {
    std::string data_dir = "./data/";
    int num_batches = 100;
    int batch_size = 100000;
    int start_batch_count = 0;
    int num_poses = 64*64*64*64;
    int num_variances = 64*64*64*64;
    int max_samples = 4000000;
    std::vector<float> min_variance = {0.0, 0.0, 0.0, 0.0, 0.0};
    std::vector<float> max_variance = {0.3, 0.3, 0.3, 0.3, 0.3};
    std::vector<float> min_pose = {0.1, 0.1, 0.0};
    std::vector<float> max_pose = {5, 5, 2*M_PI};
    std::vector<float> accuracy_bins = {0.0, 0.01, 0.1, 1.0};
    std::vector<float> bin_accuracy = {0.0001, 0.001, 0.01};
    float robot_width = 4.07;
    float robot_height = 1.74;
    float spread = 3;
};

Arguments parse_args(int argc, char** argv) {
    Arguments a;
    po::options_description desc("Allowed options");
    desc.add_options()
        ("help", "produce help message")
        ("data_dir", po::value<std::string>(), "where to store the data")
        ("num_batches,n", po::value<int>(), "number of batches")
        ("batch_size,b", po::value<int>(), "number of samples per batch")
        ("start_batch_count,s", po::value<int>(), "start value for batches")
        ("num_poses", po::value<int>(), "number of poses")
        ("num_variances", po::value<int>(), "number of variances")
        ("max_samples", po::value<int>(), "maximum number of samples for z-test")
        ("accuracy_bins", po::value<std::vector<float>>()->multitoken(), "accuracy bins e.g. [0.0001 0.001 0.01 0]")
        ("bin_accuracy", po::value<std::vector<float>>()->multitoken(), "accuracy for each bin e.g. [0.0001, 0.001, 0.01]")
        ("min_variance", po::value<std::vector<float>>()->multitoken(), "min variance for each dimension e.g. [0.0, 0.0, 0.0, 0.0, 0.0]")
        ("max_variance", po::value<std::vector<float>>()->multitoken(), "max variance for each dimension e.g. [0.3, 0.3, 0.3, 0.3, 0.3]")
        ("min_pose", po::value<std::vector<float>>()->multitoken(), "min pose for each dimension e.g. [0.1, 0.1, 0.0]")
        ("max_pose", po::value<std::vector<float>>()->multitoken(), "max pose for each dimension e.g. [5, 5, 2*M_PI]")
        ("robot_width,w", po::value<float>(), "robot width")
        ("robot_height,h", po::value<float>(), "robot height")
        ("spread", po::value<float>(), "spread of poses");
    ;

    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);    

    if (vm.count("help")) {
        std::cout << desc << "\n";
        exit(1);
    }

    if(vm.count("data_dir")) {
        a.data_dir = vm["data_dir"].as<std::string>();
    }
    if(vm.count("num_batches")) {
        a.num_batches = vm["num_batches"].as<int>();
    }
    if(vm.count("batch_size")) {
        a.batch_size = vm["batch_size"].as<int>();
    }
    if(vm.count("start_batch_count")) {
        a.start_batch_count = vm["start_batch_count"].as<int>();
    }
    if(vm.count("num_poses")) {
        a.num_poses = vm["num_poses"].as<int>();
    }
    if(vm.count("num_variances")) {
        a.num_variances = vm["num_variances"].as<int>();
    }
    if(vm.count("max_samples")) {
        a.max_samples = vm["max_samples"].as<int>();
    }
    if (vm.count("accuracy_bins")) {
        std::vector<float> values = vm["accuracy_bins"].as<std::vector<float>>();
        a.accuracy_bins = values;
    }
    if (vm.count("bin_accuracy")) {
        std::vector<float> values = vm["bin_accuracy"].as<std::vector<float>>();
        a.bin_accuracy = values;
    }
    if (vm.count("min_variance")) {
        std::vector<float> values = vm["min_variance"].as<std::vector<float>>();
        assert(values.size() == 5);
        a.min_variance = values;
    }
    if (vm.count("max_variance")) {
        std::vector<float> values = vm["max_variance"].as<std::vector<float>>();
        assert(values.size() == 5);
        a.max_variance = values;
    }
    if (vm.count("min_pose")) {
        std::vector<float> values = vm["min_pose"].as<std::vector<float>>();
        assert(values.size() == 3);
        a.min_pose = values;
    }
    if (vm.count("max_pose")) {
        std::vector<float> values = vm["max_pose"].as<std::vector<float>>();
        assert(values.size() == 3);
        a.max_pose = values;
    }
    if(vm.count("robot_width")) {
        a.robot_width = vm["robot_width"].as<float>();
    }
    if(vm.count("robot_height")) {
        a.robot_height = vm["robot_height"].as<float>();
    }
    if(vm.count("spread")) {
        a.spread = vm["spread"].as<float>();
    }
    return a;
}


#define THREADS 512


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

struct Position
{
    float x, y;
};

struct Variance
{
    float x, y, theta, width, height;
};

struct Pose
{
    float width, height, theta;
};


typedef Variance StdDev;
typedef thrust::device_vector<Position>::iterator   DeviceFloatPairIterator;
typedef thrust::device_vector<float>::iterator   DeviceFloatIterator;
typedef thrust::device_vector<int>::iterator   DeviceIntIterator;
typedef thrust::tuple<DeviceFloatPairIterator, DeviceFloatIterator, DeviceFloatIterator, DeviceFloatIterator> DeviceIteratorTuple;
typedef thrust::zip_iterator<DeviceIteratorTuple> DeviceZipIterator;


__global__ void setup_kernel(hiprandState *state, int seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(seed+id, id, 0, &state[id]);
}

__device__ __host__
void create_rect(float* r, float w, float h)
{
    r[0] = -w/2;
    r[1] = -h/2;
    r[2] = w/2;
    r[3] = -h/2;
    r[4] = w/2;
    r[5] = h/2;
    r[6] = -w/2;
    r[7] = h/2; 
}

__device__ void rot_trans_rectangle(float* r, float dx, float dy, float dt){
    float c = cosf(dt);
    float s = sinf(dt);
    float x,y;
    for(int i = 0; i < 4; i++){
        x = r[2*i];
        y = r[2*i+1];
        r[2*i] = c*x-s*y + dx;
        r[2*i+1] = s*x+c*y + dy;
    }
}

__device__ void sample_rectangle(float* r_in, float* r_out, StdDev& std_dev, hiprandState* state)
{
    float dx = hiprand_normal(state) * std_dev.x;
    float dy = hiprand_normal(state) * std_dev.y; 
    float dt = hiprand_normal(state) * std_dev.theta;
    float dw = hiprand_normal(state) * std_dev.width; 
    float dh = hiprand_normal(state) * std_dev.height;
    
    memcpy(r_out, r_in, sizeof(float) * 8);
    float dwh[8];
    create_rect(dwh, dw, dh);
    for (int i = 0; i < 8; i++){ r_out[i] += dwh[i]; }
    rot_trans_rectangle(r_out, dx, dy, dt);
}

__device__ int convex_collide(float* r1, float* r2) {
    float* rs[2] = {r1, r2};
    float norm[2];
    float p1[4];
    float p2[4];
    float* r;

    int collide = 1;
    for(int j = 0; j < 2; j++){
        r = rs[j];
        for(int i = 0; i < 4; i++){
            norm[0] = r[(i+1)*2%8] - r[i*2];
            norm[1] = r[((i+1)*2+1)%8] - r[i*2+1];
            for(int k = 0; k < 4; k++){
                p1[k] = norm[0]*r1[k*2]+norm[1]*r1[k*2+1];
                p2[k] = norm[0]*r2[k*2]+norm[1]*r2[k*2+1];
            }
            thrust::pair<float *, float *> result1 = thrust::minmax_element(thrust::device, p1, p1 + 4);
            thrust::pair<float *, float *> result2 = thrust::minmax_element(thrust::device, p2, p2 + 4);
            if(*result1.second < *result2.first || *result2.second < *result1.first){
                collide = 0;
            }
        }
    }
    return collide;
}

__device__
float calcSlack(int nsamples, int nsamples_true){
    float z = 1.96;
    float alpha = 0.025;
    if((nsamples_true == nsamples) || (nsamples_true == 0)){
        return log(1.0 / alpha) / nsamples;
    }
    else{
        return z / nsamples * sqrt((float) nsamples_true - nsamples_true * nsamples_true / (float) nsamples);
    }
}

__device__
int getBin(float p, float* accuracy_bins, int n_accuracy_bins){
    int bin = 0;
    for (int i = 0; i < n_accuracy_bins; i++){
        if(p >= accuracy_bins[i] && p <= accuracy_bins[i+1]){
            bin = i;
        }
    }
    return bin;
}


__global__ void write_collision_probability(float* n_true, int n_done, int n_samples){
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(gidx >= n_done)
        return;
    n_true[gidx] =  n_true[gidx] / (float) n_samples;
}


__global__ void monte_carlo_sample_collision_dataset_uniform(float* robot_base,
                                                                Pose* poses,
                                                                StdDev* std_devs,
                                                                float* pose_idxs,
                                                                float* std_dev_idxs,
                                                                Position* positions,
                                                                float* cps,
                                                                float* accuracy_bins,
                                                                float* bin_accuracy,
                                                                int n_accuracy_bins,
                                                                int* done,
                                                                int iteration,
                                                                int n_samples,
                                                                int n_batch,
                                                                int num_left,
                                                                int num_poses,
                                                                int num_variances,
                                                                float r_offset,
                                                                float spread,
                                                                hiprandState* state) {
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(gidx >= num_left)
        return;

    hiprandState* localState = &state[gidx];

    int pose_idx;
    int std_dev_idx;
    Position pos;
    int n_samplestrue = 0;
    Pose pose;
    StdDev std_dev;
    if(iteration == 0){
        pose_idx = hiprand(localState) % num_poses;
        std_dev_idx = hiprand(localState) % num_variances;
        pose = poses[pose_idx];
        std_dev = std_devs[std_dev_idx];

        float theta = hiprand_uniform(localState) * 2 * M_PI;
        float shift = hiprand_normal(localState) * ((std_dev.y+std_dev.x)/2)*spread;
        pos.x = cosf(theta) * ((pose.width/2+r_offset + std_dev.x) + shift); 
        pos.y = sinf(theta) * ((pose.height/2+r_offset + std_dev.y) + shift); 
        positions[gidx] = pos;
        pose_idxs[gidx] = pose_idx;
        std_dev_idxs[gidx] = std_dev_idx;    
    } else {
        n_samplestrue = (int) cps[gidx];
        pos = positions[gidx];
        pose_idx = pose_idxs[gidx];
        std_dev_idx = std_dev_idxs[gidx];
        pose = poses[pose_idx];
        std_dev = std_devs[std_dev_idx];
    }


    float obstacle[8];
    create_rect(obstacle, pose.width, pose.height);    
    float sampled_obstacle[8];
    float robot[8];

    memcpy(robot, robot_base, sizeof(float) * 8);
    rot_trans_rectangle(robot, pos.x, pos.y, pose.theta);

    for (int i = 0; i < n_batch; i++)
    {
        sample_rectangle(obstacle, sampled_obstacle, std_dev, localState);
        n_samplestrue += convex_collide(robot, sampled_obstacle);
    }
    float slack = calcSlack(n_samples, n_samplestrue);

    float p = (float) n_samplestrue / (float) n_samples;
    int d = 0;
    if(slack <= bin_accuracy[getBin(p, accuracy_bins, n_accuracy_bins)]){
        d = 1;
    }

    done[gidx] = d;
    cps[gidx] = n_samplestrue;
}

int main(int argc, char* argv[])
{   
    Arguments args = parse_args(argc, argv);
    std::string data_dir = args.data_dir;
    int num_batches = args.num_batches;
    int batch_size = args.batch_size;
    int start_batch_count = args.start_batch_count;
    int num_poses = args.num_poses;
    int num_variances = args.num_variances;
    int pose_dim = 3;
    int variance_dim = 5;
    std::vector<float>& accuracy_bins = args.accuracy_bins;
    std::vector<float>& bin_accuracy = args.bin_accuracy;

    
    std::cout << "data dir: " << data_dir << std::endl;
    std::cout << "num batches: " << num_batches << std::endl;
    std::cout << "num batch: " << batch_size << std::endl;
    std::cout << "start batch count: " << start_batch_count << std::endl;

    // write_config(data_dir, batch_size, num_batches);

    Pose* poses = (Pose*) malloc(sizeof(Pose)*num_poses);
    StdDev* std_devs = (Variance*) malloc(sizeof(StdDev)*num_variances);  
    std::vector<Variance> variances(num_variances);  

    std::default_random_engine generator;
    std::vector<std::uniform_real_distribution<float>> pose_uniforms;
    std::vector<std::uniform_real_distribution<float>> variance_uniforms;

    for(int i = 0; i < pose_dim; i++){
        pose_uniforms.push_back(std::uniform_real_distribution<float>(args.min_pose[i], args.max_pose[i]));
    }
    for(int i = 0; i < variance_dim; i++){
        variance_uniforms.push_back(std::uniform_real_distribution<float>(args.min_variance[i], args.max_variance[i]));
    }

    for (int i = 0; i < num_poses; i++)
    {
        poses[i].width = pose_uniforms[0](generator);
        poses[i].height = pose_uniforms[1](generator);
        poses[i].theta = pose_uniforms[3](generator);
    }    
    for (int i = 0; i < num_variances; i++)
    {
        variances[i].x = variance_uniforms[0](generator);
        variances[i].y = variance_uniforms[1](generator);
        variances[i].theta = variance_uniforms[3](generator);
        variances[i].width = variance_uniforms[4](generator);
        variances[i].height = variance_uniforms[5](generator);     
        std_devs[i].x = sqrt(variances[i].x);
        std_devs[i].y = sqrt(variances[i].y);
        std_devs[i].theta = sqrt(variances[i].theta);
        std_devs[i].width = sqrt(variances[i].width);   
        std_devs[i].height = sqrt(variances[i].height);   
    }

    // write poses and variances
    size_t poses_shape[2] = {(size_t) num_poses, (size_t) 3};
    size_t variances_shape[2] = {(size_t) num_variances, (size_t) 5};
    npy::SaveArrayAsNumpy(data_dir + std::string("/poses.npy"), false, 2, poses_shape, (float*) poses);
    npy::SaveArrayAsNumpy(data_dir + std::string("/variances.npy"), false, 2, variances_shape, (float*) variances.data());

    struct stat st = {0};

    if (stat(data_dir.c_str(), &st) == -1) {
        mkdir(data_dir.c_str(), 0700);
    }
    if(stat((data_dir + "/meta").c_str(), &st) == -1){
        mkdir((data_dir + "/meta").c_str(), 0700);
    }
    size_t accuracy_bins_shape[1] = {(size_t) accuracy_bins.size()};
    size_t bin_accuracy_shape[1] = {(size_t) bin_accuracy.size()};
    npy::SaveArrayAsNumpy(data_dir + "/meta" + std::string("/accuracy_bins.npy"), false, 1, accuracy_bins_shape, accuracy_bins);
    npy::SaveArrayAsNumpy(data_dir + "/meta" + std::string("/bin_accuracy.npy"), false, 1, bin_accuracy_shape, bin_accuracy);

    float* robot = (float*) malloc(sizeof(float)*4*2);
    Position* positions = (Position*) malloc(sizeof(Position) * batch_size);
    float* pose_idxs = (float*) malloc(sizeof(float) * batch_size);
    float* var_idxs = (float*) malloc(sizeof(float) * batch_size);
    float* cp = (float*) malloc(sizeof(float) * batch_size);
    
    float* d_accuracy_bins;
    float* d_bin_accuracy;
    float* d_robot; 
    Pose* d_poses; 
    StdDev* d_std_devs; 
    Position* d_positions; 
    float* d_pose_idxs; 
    float* d_var_idxs; 
    float* d_cp; 
    int* d_done; 

    hipMalloc(&d_accuracy_bins, sizeof(float)*(accuracy_bins.size()));
    hipMalloc(&d_bin_accuracy, sizeof(float)*(bin_accuracy.size()));
    hipMalloc(&d_robot, sizeof(float)*(4*2));
    hipMalloc(&d_poses, sizeof(Pose)*num_poses);
    hipMalloc(&d_std_devs, sizeof(StdDev)*num_variances);
    hipMalloc(&d_positions, sizeof(Position)*batch_size);
    hipMalloc(&d_pose_idxs, sizeof(float)*(batch_size));
    hipMalloc(&d_var_idxs, sizeof(float)*(batch_size));
    hipMalloc(&d_cp, sizeof(float)*(batch_size));
    hipMalloc(&d_done, sizeof(int)*(batch_size));

    DeviceZipIterator d_iter(thrust::make_tuple(thrust::device_pointer_cast(d_positions), 
                                        thrust::device_pointer_cast(d_cp),
                                        thrust::device_pointer_cast(d_var_idxs),
                                        thrust::device_pointer_cast(d_pose_idxs)));


    std::vector<float> dataset(batch_size*5);

    hiprandState *devStates;
    hipMalloc((void **)&devStates, batch_size *  sizeof(hiprandState));

    dim3 threadsPerBlock(THREADS);
    dim3 numBlocks((int) std::max(1.0, ceil(batch_size/threadsPerBlock.x)));  

    // Initialize array
    create_rect(robot, args.robot_width, args.robot_height);
    float r_offset = (args.robot_width+args.robot_height)/4;

    // Transfer data from host to device memory
    hipMemcpy(d_robot, robot, sizeof(float)*(4*2), hipMemcpyHostToDevice);
    hipMemcpy(d_poses, poses, sizeof(Pose)*num_poses, hipMemcpyHostToDevice);
    hipMemcpy(d_std_devs, std_devs, sizeof(Variance)*num_variances, hipMemcpyHostToDevice);
    hipMemcpy(d_accuracy_bins, accuracy_bins.data(), sizeof(float)*(accuracy_bins.size()), hipMemcpyHostToDevice);
    hipMemcpy(d_bin_accuracy, bin_accuracy.data(), sizeof(float)*(bin_accuracy.size()), hipMemcpyHostToDevice);
    // std::srand(std::time(0));
    CUDA_CALL(hipPeekAtLastError());

    setup_kernel<<<numBlocks, THREADS>>>(devStates, std::rand());
    CUDA_CALL(hipPeekAtLastError());

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    std::cout << "Total number of configurations: " << batch_size * num_batches << std::endl;
    std::cout << "Begin computation..." << std::endl;
    int counter = 0;
    printf("batches generated: %i/%i", counter, num_batches);

    for (int batch_index = 0; batch_index < num_batches; batch_index++)
    {
        int num_left = batch_size;
        int batch_done = 0;
        int iteration = 0;
        int n_samples = 0;
        int n_batch = 0;
        while(num_left > 0 && n_samples < args.max_samples){
            numBlocks = ((int) ceil((float) num_left/threadsPerBlock.x));  
            if(n_samples < 20000)
                n_batch = 1000;
            else
                n_batch = 100000;
            n_samples += n_batch;
            monte_carlo_sample_collision_dataset_uniform<<<numBlocks, threadsPerBlock>>>(
                d_robot,
                d_poses,
                d_std_devs,
                d_pose_idxs,
                d_var_idxs,
                d_positions,
                d_cp,
                d_accuracy_bins,
                d_bin_accuracy,
                accuracy_bins.size(),
                d_done,
                iteration,
                n_samples,
                n_batch,
                num_left,
                num_poses,
                num_variances,
                r_offset,
                args.spread,
                devStates
            );
            gpuErrchk( hipPeekAtLastError() );
            batch_done = thrust::count(thrust::device, thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + num_left), 1);
            if(batch_done > 0){
                thrust::sort_by_key(thrust::device_pointer_cast(d_done), thrust::device_pointer_cast(d_done + num_left), d_iter);
                num_left -= batch_done;
                numBlocks = ((int) ceil((float) batch_done/threadsPerBlock.x));  
                write_collision_probability<<<numBlocks, threadsPerBlock>>>(d_cp + num_left, batch_done, n_samples);
                hipMemcpy(positions + num_left, d_positions + num_left, sizeof(Position) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(cp + num_left, d_cp + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(var_idxs + num_left, d_var_idxs + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
                hipMemcpy(pose_idxs + num_left, d_pose_idxs + num_left, sizeof(float) * batch_done, hipMemcpyDeviceToHost);
            }
            iteration++;
            // printf("num left %i, n_samples %i\n", num_left, n_samples);
        }

        if(num_left > 0){
            printf("copying remaining %i over (0, %i)\n", num_left, num_left);
            numBlocks = ((int) ceil((float) num_left/threadsPerBlock.x));  
            write_collision_probability<<<numBlocks, threadsPerBlock>>>(d_cp, num_left, n_samples);
            gpuErrchk( hipPeekAtLastError() );
            hipMemcpy(positions, d_positions, sizeof(Position) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(cp, d_cp, sizeof(float) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(var_idxs, d_var_idxs, sizeof(float) * num_left, hipMemcpyDeviceToHost);
            hipMemcpy(pose_idxs, d_pose_idxs, sizeof(float) * num_left, hipMemcpyDeviceToHost); 
        }


        CUDA_CALL(hipDeviceSynchronize());

        // write data
        float* d = dataset.data();
        float* c = cp;

        for (int j = 0; j < batch_size; j++)
        {
            d[0] = positions[j].x;  // x
            d[1] = positions[j].y;  // y
            d[2] = *c;  // cp
            d[3] = var_idxs[j]; // var_idx
            d[4] = pose_idxs[j]; // pose_idx
            c+=1;
            d+=5;
        }

        // write dataset
        size_t ds_shape[2] = {(size_t) batch_size, (size_t) 5};

        npy::SaveArrayAsNumpy(data_dir + std::string("/") + std::to_string(start_batch_count + batch_index) + ".npy", false, 2, ds_shape, dataset);
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        printf("\33[2K\r");
        printf("batches generated: %i/%i, Time: %i [min]", ++counter, num_batches, (int) std::chrono::duration_cast<std::chrono::minutes>(end - begin).count());
        fflush(stdout); 
    }
    std::cout << std::endl;
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Finished computation" << std::endl;
    std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::minutes>(end - begin).count() << " [min]" << std::endl;

    // free memory
    hipFree(devStates);
    hipFree(d_accuracy_bins);
    hipFree(d_bin_accuracy);
    hipFree(d_robot);
    hipFree(d_poses);
    hipFree(d_std_devs);
    hipFree(d_positions);
    hipFree(d_pose_idxs);
    hipFree(d_var_idxs);
    hipFree(d_cp);
    hipFree(d_done);
    std::cout << "Done." << std::endl;
}
